/*
 * Newton-Raphson Method CUDA Implementation
 * 
 * This program implements the Newton-Raphson root-finding algorithm on a GPU using CUDA.
 * The implementation allows finding multiple roots in parallel by starting from different
 * initial guesses distributed across a range.
 *
 * The Newton-Raphson method iteratively refines an initial guess using the formula:
 *    x_{n+1} = x_n - f(x_n)/f'(x_n)
 * 
 * The algorithm converges quadratically when near a root, making it very efficient
 * for finding roots of differentiable functions.
 * The comments explain:
 * The overall purpose and mathematical foundation of the algorithm
 * Each function's purpose and parameters
 * GPU-specific optimizations and considerations
 * Numerical stability safeguards
 * Memory management strategy
 * The parallel approach to finding multiple roots simultaneously
 
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

// Define a structure to contain function and derivative code
// This allows passing function pointers to device code for more flexibility
typedef struct {
    float (*f)(float);   // Function pointer for f(x)
    float (*df)(float);  // Function pointer for the derivative f'(x)
} FunctionPair;

// Sample function: f(x) = x^3 - 2x^2 + 4x - 8
// The __device__ qualifier makes this function callable from the device (GPU)
__device__ float f(float x) {
    return x*x*x - 2*x*x + 4*x - 8;  // Cubic polynomial with potentially multiple roots
}

// Its derivative: f'(x) = 3x^2 - 4x + 4
// Analytical derivative for faster and more accurate convergence
__device__ float df(float x) {
    return 3*x*x - 4*x + 4;  // Quadratic polynomial (derivative of cubic)
}

/*
 * Newton-Raphson kernel with hard-coded function
 * 
 * Parameters:
 * - x0: Array of initial guesses
 * - result: Array to store computed roots
 * - tolerance: Convergence criterion (stop when |x_{n+1} - x_n| < tolerance)
 * - maxIterations: Maximum number of iterations to prevent infinite loops
 */
__global__ void newtonRaphsonKernel(float *x0, float *result, float tolerance, int maxIterations) {
    // Calculate global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load the initial guess for this thread
    float x = x0[idx];
    float x_new;
    float error = tolerance + 1.0f;  // Initialize error to be > tolerance
    int iter = 0;
    
    // Iterate until convergence or max iterations reached
    while (error > tolerance && iter < maxIterations) {
        // Newton-Raphson formula: x_{n+1} = x_n - f(x_n)/f'(x_n)
        float fx = f(x);      // Calculate function value
        float dfx = df(x);    // Calculate derivative value
        
        // Check for division by zero (avoid numerical instability)
        if (fabsf(dfx) < 1e-10f) {
            // If derivative is too small, iteration might diverge
            break;
        }
        
        // Calculate the next approximation
        x_new = x - fx / dfx;
        
        // Calculate error for convergence check
        error = fabsf(x_new - x);
        
        // Update current approximation
        x = x_new;
        iter++;
    }
    
    // Store the result for this thread
    result[idx] = x;
}

/*
 * Generic Newton-Raphson kernel that uses function pointers
 * This version allows changing the target function without recompiling
 * 
 * Parameters:
 * - Same as above, plus:
 * - funcs: Structure containing function and derivative pointers
 */
__global__ void newtonRaphsonGeneric(float *x0, float *result, float tolerance, 
                                    int maxIterations, FunctionPair funcs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    float x = x0[idx];
    float x_new;
    float error = tolerance + 1.0f;
    int iter = 0;
    
    while (error > tolerance && iter < maxIterations) {
        // Use function pointers instead of hard-coded functions
        float fx = funcs.f(x);
        float dfx = funcs.df(x);
        
        if (fabsf(dfx) < 1e-10f) {
            break;
        }
        
        x_new = x - fx / dfx;
        error = fabsf(x_new - x);
        x = x_new;
        iter++;
    }
    
    result[idx] = x;
}

/*
 * Main function:
 * - Sets up the problem
 * - Allocates memory on host and device
 * - Launches kernel
 * - Processes and displays results
 */
int main() {
    // Number of parallel attempts (different initial guesses)
    const int N = 256;
    
    // Host memory for initial guesses and results
    float h_x0[N];
    float h_result[N];
    
    // Create initial guesses spread over a range
    // This increases the likelihood of finding all roots of the function
    for (int i = 0; i < N; i++) {
        h_x0[i] = -10.0f + 20.0f * i / (N - 1); // Spread from -10 to 10
    }
    
    // Allocate device memory for input and output
    float *d_x0, *d_result;
    hipMalloc(&d_x0, N * sizeof(float));
    hipMalloc(&d_result, N * sizeof(float));
    
    // Copy the initial guesses to the device
    hipMemcpy(d_x0, h_x0, N * sizeof(float), hipMemcpyHostToDevice);
    
    // Set up block and grid dimensions for CUDA kernel
    // For optimal performance, threadsPerBlock should be a multiple of 32 (warp size)
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    // Launch the Newton-Raphson kernel
    // Parameters: tolerance=1e-6 (6 digits precision), maxIterations=100
    newtonRaphsonKernel<<<blocksPerGrid, threadsPerBlock>>>(d_x0, d_result, 1e-6f, 100);
    
    // Copy the results back to the host
    hipMemcpy(h_result, d_result, N * sizeof(float), hipMemcpyDeviceToHost);
    
    // Print the results
    printf("Found roots (might include duplicates):\n");
    for (int i = 0; i < N; i++) {
        // Check if the value is actually a root by evaluating the function
        // This filters out cases where the method failed to converge
        float fx = h_result[i]*h_result[i]*h_result[i] - 
                  2*h_result[i]*h_result[i] + 
                  4*h_result[i] - 8;
        
        // Only print values where f(x) is very close to zero
        if (fabsf(fx) < 1e-4f) {
            printf("Root at x = %.8f (f(x) = %.8e)\n", h_result[i], fx);
        }
    }
    
    // Free device memory to prevent memory leaks
    hipFree(d_x0);
    hipFree(d_result);
    
    return 0;
}
